﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/*input size: 131072*/

// __device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
//                              int input_size) {
//   for (int i = 0; i < input_size; i++)
//     vm_write(vm, i, input[i]);

//   for (int i = input_size - 1; i >= input_size - 32769; i--)
//     int value = vm_read(vm, i);

//   vm_snapshot(vm, results, 0, input_size);
// }
// expect page fault num: 8193



__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
  int input_size) {
  // write the data.bin to the VM starting from address 32*1024
  for (int i = 0; i < input_size; i++)
    vm_write(vm, 32*1024+i, input[i]);
  // write (32KB-32B) data  to the VM starting from 0
  for (int i = 0; i < 32*1023; i++)
    vm_write(vm, i, input[i+32*1024]);
  // readout VM[32K, 160K] and output to snapshot.bin, which should be the same with data.bin
  vm_snapshot(vm, results, 32*1024, input_size);
}

// expected page fault num: 9215

